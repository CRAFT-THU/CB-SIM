#include "hip/hip_runtime.h"
#include "crossbar_cuda.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <time.h>
#include <math.h>

__global__ void CUDA_abs(float *a,float *b,int cols,int rows) {
    int n_cell= blockIdx.x ;
    int row  = blockIdx.y ;
    int col = threadIdx.x;
    b[n_cell*rows*cols+row*cols+col]=fabs(a[n_cell*rows*cols+row*cols+col]);
}

__global__ void CUDA_add(float *a,float *b,float *c,int cols,int rows) {
    int n_cell= blockIdx.x ;
    int row  = blockIdx.y ;
    int col = threadIdx.x;
    c[n_cell*rows*cols+row*cols+col]=a[n_cell*rows*cols+row*cols+col]+b[n_cell*rows*cols+row*cols+col];
}

__global__ void CUDA_mul(float *a,float b,float *c,int cols,int rows) {
    int n_cell= blockIdx.x ;
    int row  = blockIdx.y ;
    int col = threadIdx.x;
    c[n_cell*rows*cols+row*cols+col]=a[n_cell*rows*cols+row*cols+col]*b;
}


__global__ void CUDA_mmul(float *a,float *b,float *c,int cols,int rows) {
    int n_cell= blockIdx.x ;
    int row  = blockIdx.y ;
    int col = threadIdx.x;
    c[n_cell*rows*cols+row*cols+col]=a[n_cell*rows*cols+row*cols+col]*b[n_cell*rows*cols+row*cols+col];
}

__global__ void CUDA_shift(float *a,float b,float *c,int cols,int rows) {
    int n_cell= blockIdx.x ;
    int row  = blockIdx.y ;
    int col = threadIdx.x;
    c[n_cell*rows*cols+row*cols+col]=a[n_cell*rows*cols+row*cols+col]+b;
}

__global__ void CUDA_MatrixMui(float *a,float *b,float *c,int cols,int rows) {
    int n_cell= blockIdx.x ;
    int row  = blockIdx.y ;
    //int col = threadIdx.x;
    float temp = 0;
    for (int i=0;i<cols;i++)
    {
        temp+=a[n_cell*cols+i]*b[n_cell*rows*cols+row*cols+i];

    }

    c[n_cell*rows+row]=temp;
}

void Crossbar::init(float *CB_cells, int n, int l, int w)
{
    CB_l=l;
    CB_w=w;
    CB_n=n;
    hipMalloc((void **)&CB_cell, CB_n*CB_l*CB_w*sizeof(float));
    hipMemcpy(CB_cell, CB_cells, CB_n*CB_l*CB_w * sizeof(float),hipMemcpyHostToDevice);
    get_std();
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    clock_t time;
    time=clock();
    hiprandSetPseudoRandomGeneratorSeed(gen, (int)time);

}

void Crossbar::printcrossbar() {
    float *temp_cell;
    temp_cell = (float*)calloc(CB_n*CB_l*CB_w,sizeof(float));
    //temp_cell = new float [CB_n*CB_l*CB_w];
    hipMemcpy(temp_cell, CB_cell, CB_n*CB_l*CB_w* sizeof(float),hipMemcpyDeviceToHost) ;
    printf ("_______________\n");
    for (int i=0;i<CB_n;i++)
    {
        for (int j=0;j<CB_l;j++)
        {
            for(int k=0;k<CB_w;k++)
            {
                printf("%f,%d,%d,%d,%d ",temp_cell[i*CB_l*CB_w+j*CB_w+k],i,j,k,i*CB_l*CB_w+j*CB_w+k);

            }
            printf ("\n");
        }
        printf ("\n");
    }
    printf ("_______________\n");
    free(temp_cell);
}

void Crossbar::get_std() {//-0.0006034 * (x * 1e3) ** 2 + 0.06184 * x + 0.7240 * 1e-6
    dim3 numBlocks(CB_n, CB_l);
    hipMalloc((void **)&std_d, CB_n*CB_l*CB_w*sizeof(float));
    float *temp_1;
    hipMalloc((void **)&temp_1, CB_n*CB_l*CB_w*sizeof(float));
    //hipMemcpy(temp_1, CB_cell, CB_n*CB_l*CB_w* sizeof(float),hipMemcpyDeviceToDevice) ;
    CUDA_abs<<<numBlocks,CB_w>>>(CB_cell,temp_1,CB_w,CB_l);
    float *temp_2;
    hipMalloc((void **)&temp_2, CB_n*CB_l*CB_w*sizeof(float));
    CUDA_mul<<<numBlocks,CB_l>>>(temp_1,1000,temp_2,CB_w,CB_l);
    hipMemcpy(temp_1, temp_2, CB_n*CB_l*CB_w* sizeof(float),hipMemcpyDeviceToDevice) ;
    float *temp_3;
    hipMalloc((void **)&temp_3, CB_n*CB_l*CB_w*sizeof(float));
    CUDA_mmul<<<numBlocks,CB_w>>>(temp_1,temp_2,temp_3,CB_w,CB_l);
    CUDA_mul<<<numBlocks,CB_w>>>(temp_3,-0.0006034,temp_1,CB_w,CB_l);
    CUDA_mul<<<numBlocks,CB_w>>>(CB_cell,0.06184,temp_2,CB_w,CB_l);
    CUDA_add<<<numBlocks,CB_w>>>(temp_1,temp_2,temp_3,CB_w,CB_l);
    CUDA_shift<<<numBlocks,CB_w>>>(temp_3,0.7240*0.000001,temp_1,CB_w,CB_l);
    hipMemcpy(std_d, temp_1, CB_n*CB_l*CB_w* sizeof(float),hipMemcpyDeviceToDevice) ;
    hipFree( temp_1 );
    hipFree( temp_2 );
    hipFree( temp_3 );
}

void Crossbar::get_noise(float *noise) {
    hipMalloc((void **)&noise, CB_n*CB_l*CB_w*sizeof(float));
    hiprandGenerateNormal(gen, noise, CB_n*CB_l*CB_w, 0, 1);
}

void Crossbar::printstd() {
    printf ("~~~~~~~~~~~~~~~~~~~\n");
    float *temp_cell;
    temp_cell = (float*)calloc(CB_n*CB_l*CB_w,sizeof(float));
    hipMemcpy(temp_cell, std_d, CB_n*CB_l*CB_w* sizeof(float),hipMemcpyDeviceToHost) ;
    for (int i=0;i<CB_n;i++)
    {
        for (int j=0;j<CB_l;j++)
        {
            for(int p=0;p<CB_w;p++)
            {
                printf("%f ",temp_cell[i*CB_l*CB_w+j*CB_w+p]);
            }
            printf ("\n");
        }
        printf ("\n");
    }
    printf ("~~~~~~~~~~~~~~~~~~~\n");
    free(temp_cell);
}

void Crossbar::run(float *input, float *output, bool use_noise=true) {
    float *input_d,*output_d;
    hipMalloc((void **)&input_d, CB_n*CB_w*sizeof(float));
    hipMalloc((void **)&output_d, CB_n*CB_l*sizeof(float));
    hipMemcpy(input_d, input, CB_n*CB_w * sizeof(float),hipMemcpyHostToDevice);
    dim3 numBlocks(CB_n, CB_l);
    if (use_noise)
    {
        float *temp_noise,*temp_cell;
        hipMalloc((void **)&temp_noise, CB_n*CB_w*CB_l*sizeof(float));
        hipMalloc((void **)&temp_cell, CB_n*CB_w*CB_l*sizeof(float));
        get_noise(temp_noise);
        CUDA_add<<<numBlocks,CB_w>>>(CB_cell,temp_noise,temp_cell,CB_w,CB_l);
        CUDA_MatrixMui<<<numBlocks,1>>>(input_d,temp_cell,output_d,CB_w,CB_l);
    }
    else
    {
        CUDA_MatrixMui<<<numBlocks,1>>>(input_d,CB_cell,output_d,CB_w,CB_l);
    }
    hipMemcpy(output, output_d, CB_n*CB_l* sizeof(float),hipMemcpyDeviceToHost) ;
    hipFree( input_d );
    hipFree( output_d );
}

void Crossbar::free_space() {
    // do something
}